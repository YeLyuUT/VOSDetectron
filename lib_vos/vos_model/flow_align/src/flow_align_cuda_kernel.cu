#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C" {
#endif

#include <stdio.h>
#include <math.h>
#include <float.h>
#include "flow_align_cuda_kernel.h"

#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
    i += blockDim.x * gridDim.x)


__global__ void FlowAlignForward_kernel(const int nthreads,const  int height, const int width, const int channels, const double* bottom, const double* flow,double* top)
{

  CUDA_1D_KERNEL_LOOP(index, nthreads)
  {
      // get feature position
      int w = index % width;
      int h = (index / width) %  height;
      int c  = (index / width /  height) % channels;
      int n  = index / width /  height / channels;
      // get flow
      int ind_flow_x = w+h*width+n*height*width*2;
      int ind_flow_y = w+h*width+width*height+n*height*width*2;
      double flo_x = flow[ind_flow_x];
      double flo_y = flow[ind_flow_y];
      // get bilinear positions
      double w_flo = w+flo_x;
      double h_flo = h+flo_y;
      if ( h_flo<0 || h_flo>height-1 || w_flo<0 || w_flo>width-1 )
      {
        //outof image, we set it to 0.
        top[index] = 0;
      }
      else{
        int h_start = floor(h_flo);
        int w_start = floor(w_flo);
        int nc_start = n*height*width*channels+c*height*width;
        double h_ratio = h_flo - (double)h_start;
        double w_ratio = w_flo - (double)w_start;
        int upleft = nc_start+w_start+width*h_start;
        int upright = upleft+1;
        int downleft = upleft+width;
        int downright = downleft+1;

        top[index] = bottom[upleft] * (1.-h_ratio) * (1.-w_ratio)
                         + bottom[upright] * (1.-h_ratio) * (w_ratio)
                         + bottom[downleft] * (h_ratio) * (1.-w_ratio)
                         + bottom[downright] * (h_ratio) * (w_ratio);
      }
  }
}

__global__ void FlowAlignBackward_kernel(const int nthreads, const int height, const int width, const int channels, const double* topdiff,const double* bottom, const double*flow, double* bottomdiff, double* flowdiff)
{
  CUDA_1D_KERNEL_LOOP(index, nthreads)
  {
      // get feature position
      int w = index % width;
      int h = (index / width) %  height;
      int c  = (index / width /  height) % channels;
      int n  = index / width /  height / channels;
      // get flow
      int ind_flow_x = w+h*width+n*height*width*2;
      int ind_flow_y = w+h*width+width*height+n*height*width*2;
      double flo_x = flow[ind_flow_x];
      double flo_y = flow[ind_flow_y];
      // get bilinear positions
      double w_flo = w+flo_x;
      double h_flo = h+flo_y;
      if ( h_flo<0 || h_flo>height-1 || w_flo<0 || w_flo>width-1 )
      {
        //outof image, no grad propagated.
        continue;
      }
      else
      {
        int h_start = floor(h_flo);
        int w_start = floor(w_flo);
        int nc_start = n*height*width*channels+c*height*width;
        double h_ratio = h_flo - (double)h_start;
        double w_ratio = w_flo - (double)w_start;
        int upleft = nc_start+w_start+width*h_start;
        int upright = upleft+1;
        int downleft = upleft+width;
        int downright = downleft+1;

        //bottom diff
        atomicAdd(bottomdiff+upleft, topdiff[index] * (1.-h_ratio) * (1.-w_ratio));
        atomicAdd(bottomdiff+upright, topdiff[index] *  (1.-h_ratio) * (w_ratio));
        atomicAdd(bottomdiff+downleft, topdiff[index] * (h_ratio) * (1.-w_ratio));
        atomicAdd(bottomdiff+downright, topdiff[index] * (h_ratio) * (w_ratio));

        //approximate flow diff
        double f1 = bottom[upleft];
        double f2 = bottom[upright];
        double f3 = bottom[downleft];
        double f4 = bottom[downright];
        //recalculate top rather than saving it temporarily in the layer.
        /*
        top[index] = bottom[upleft] * (1.-h_ratio) * (1.-w_ratio)
                         + bottom[upright] * (1.-h_ratio) * (w_ratio)
                         + bottom[downleft] * (h_ratio) * (1.-w_ratio)
                         + bottom[downright] * (h_ratio) * (w_ratio);
                         */        
        //calculate dx
        double dx = -f1*(1.-h_ratio) + f2*(1.-h_ratio) - f3*(h_ratio) + f4*(h_ratio);
        //calculate dy
        double dy = -f1*(1.-w_ratio) - f2*(w_ratio) + f3*(1.-w_ratio) + f4*(w_ratio);
        atomicAdd(flowdiff+ind_flow_x, topdiff[index]*dx);
        atomicAdd(flowdiff+ind_flow_y, topdiff[index]*dy);
      }
  }
}

int FlowAlignForward(const int batches, const int height, const int width, const int channels, const double* bottom, const double* flow,double* top, hipStream_t stream)
{
  const int kThreadPerBlock = 512;
  const int nthreads = batches*height*width*channels;
  hipError_t err;

  FlowAlignForward_kernel<<<(nthreads+kThreadPerBlock-1)/kThreadPerBlock, kThreadPerBlock, 0, stream>>>(
    nthreads, height, width, channels, bottom, flow, top);

  err = hipGetLastError();
  if(hipSuccess != err) {
      fprintf( stderr, "cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
      exit( -1 );
  }

  return 1;
}


int FlowAlignBackward(const int batches, const int height, const int width, const int channels, const double* topdiff, const double*bottom, const double*flow, double* bottomdiff, double* flowdiff, hipStream_t stream)
{
  const int kThreadPerBlock = 512;
  const int nthreads = batches*height*width*channels;
  hipError_t err;
  FlowAlignBackward_kernel<<<(nthreads+kThreadPerBlock-1)/kThreadPerBlock, kThreadPerBlock, 0, stream>>>(
    nthreads, height, width, channels, topdiff, bottom, flow, bottomdiff, flowdiff);

  err = hipGetLastError();
  if(hipSuccess != err) {
      fprintf( stderr, "cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
      exit( -1 );
  }

  return 1;
}


#ifdef __cplusplus
}
#endif
